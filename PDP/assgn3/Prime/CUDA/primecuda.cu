#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>
 
#define NUM_BLOCKS 1024

__global__ void primecuda( int *dev_num, int *arr) 
{
	int blckid = blockIdx.x;	//Handle the data at the block index
	int tid = threadIdx.x;
	int gid = 1+(*dev_num/1024);
	int bid = (blckid*gid) + tid;
	int flag, i;
	flag = 0;
	if(bid <= *dev_num)
	{
		if(bid % 2 != 0)			
		{
			for (i = 3 ; i<= sqrtf(bid); i++)
			{
				if(bid % i == 0)
				{
					flag = 1;
					break;
				}
			}
			if (flag == 0)
			{
				if (bid == 0 || bid == 1 || bid == 2)
				{
					//Do nothing
				}
				else
				{
					arr[bid] = bid;
				}
			}
		}
	}		
}

int main(int argc, char **argv)
{
	if(strcmp("-t",argv[1])!=0)
	{
		printf("Error\n");
		return 0;
	}
	
	int *dev_num;
	int num,count,biggest;
	num = atoi(argv[2]);
	int i;
	int *dev_arr;
	int arr[num];
	int NUM_THREADS;
	clock_t begin, end;
	
	begin = clock();
	
	for(i = 0;i<num;i++)
	{
		arr[i] = 0;
		
	}
	NUM_THREADS = 1+(num/NUM_BLOCKS);
	count = 1;
	biggest = 2;
	
	hipMalloc ( (void**)&dev_num, sizeof (int) );
	hipMalloc ( (void**)&dev_arr, num * sizeof (int) );
	
	hipMemcpy( dev_num, &num, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy( dev_arr, arr, num * sizeof(int), hipMemcpyHostToDevice);
	
	primecuda<<<NUM_BLOCKS, NUM_THREADS>>> (dev_num, dev_arr);
	
	hipMemcpy( arr, dev_arr, num * sizeof(int), hipMemcpyDeviceToHost);
	
	
	
	for(i = 0; i<num;i++)
	{
		if(arr[i]> 0)
		{
			//printf("%d	",arr[i]);
			count++;
			if(arr[i]>biggest)
				biggest = arr[i];
		}
	}
	end = clock();
	printf("The largest prime number is: %d\n",biggest);
	printf("The number of prime numbers are: %d\n",count);
	printf("The time is: %f\n", (double)(end - begin) / CLOCKS_PER_SEC);
	
	hipFree( dev_num );
	hipFree( dev_arr );
	
	return 0;
}