#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>
#include<math.h>
#include<time.h>
#include<stdint.h>
#include<string.h>
#include"ziggurat.h"
#include"rnd.h"
#include"ziggurat.c"
#include"rnd.c"

#define NUM_THREADS 8

__global__ void sortcuda( float *arr, int *num, float *bucket)
{
	int size = *num;
	int tid = threadIdx.x;
	int range,i;
	int count = 0;
	int localcount;
	int val = tid*size;
	
	range = (1+(*num/NUM_THREADS));
	
	for(i=0 ;i<*num ; i++)
	{
		if(floor(arr[i]/range) == tid)
		{
			localcount = count;
			while(localcount > 0 && bucket[val+localcount -1] > arr[i] )
			{
				bucket[val+localcount] = bucket[val+localcount -1];
				localcount--;
			}
			bucket[val+localcount]=arr[i];
			count++;
		}
	}
}

int main(int argc, char **argv)
{
	clock_t begin, end;
	begin = clock();
	if(strcmp("-t",argv[1])!=0)
	{
		printf("Error\n");
		return 0;
	}
	int num,max_number;
	//int j;float temp;
	int i;
	num = atoi(argv[2]);
	max_number = atoi(argv[2]);
	float bucket[num*NUM_THREADS];
	float arr[num];
	float arr1[num];
	
	for(i = 0;i<num;i++)
	{
		bucket[i] = 0;
	}

	float *dev_bucket;
	float *dev_arr;
	int *dev_num;

	random_number_generator_simple(arr, num, max_number);
	random_number_generator_simple(arr1, num, max_number);	

	hipMalloc ((void**)&dev_arr, num*sizeof(float));
	hipMalloc ((void**)&dev_bucket, num*NUM_THREADS*sizeof(float));
	hipMalloc ((void**)&dev_num, sizeof (int));
	
	
	hipMemcpy( dev_arr, arr, num*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy( dev_bucket, bucket, num*NUM_THREADS* sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy( dev_num, &num, sizeof(int), hipMemcpyHostToDevice);
	
	sortcuda<<<1,NUM_THREADS>>>(dev_arr,dev_num,dev_bucket);
	
	hipMemcpy( bucket, dev_bucket, num*NUM_THREADS* sizeof(float), hipMemcpyDeviceToHost);
	
	end = clock();
	printf("\nAll Values Sorted\n");
	printf("The time is: %f\n", (double)(end - begin) / CLOCKS_PER_SEC);

	hipFree( dev_num );
	hipFree( dev_arr );
	hipFree( dev_bucket );
	return 0;
}
